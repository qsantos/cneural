#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>

extern "C"
{
#include "mnist.h"
}

#define n_inputs (28*28)
#define n_hidden (300)
#define n_outputs (10)

__device__ float weights0[n_inputs+1][n_hidden]; // inverted indices (striding)
__shared__ float weights1[n_outputs][n_hidden+1];
__shared__ float local_fields[n_hidden];
__shared__ float intermediates[n_hidden];

__device__ float sigmoid(float x)
{
    return 1.f / (1.f + expf(-0.007f * x));
}

__device__ float sigmoid_prime(float x)
{
    float s = sigmoid(x);
    return (1 - s) * s;
}

__device__ void compute(float* inputs, float* outputs)
{
    const int i = threadIdx.x;

    if (i < n_hidden)
    {
        // compute local field, v_i = sum(y_j w_ji)
        float local_field = weights0[n_inputs][i];
        for (size_t j = 0; j < n_inputs; j++)
            local_field += inputs[j] * weights0[j][i];
        local_fields[i] = local_field;

        // compute outputs, y_i = ϕ(v_i)
        intermediates[i] = sigmoid(local_field);
    }

    if (i < n_outputs)
    {
        // compute local field, v_i = sum(y_j w_ji)
        float local_field = weights1[i][n_hidden];
        for (size_t j = 0; j < n_hidden; j++)
            local_field += intermediates[j] * weights1[i][j];

        // compute outputs, y_i = ϕ(v_i)
        outputs[i] = sigmoid(local_field);
    }
}

__device__ void train(float* inputs, float* expect)
{
    const int i = threadIdx.x;

    __shared__ float outputs[n_outputs];
    compute(inputs, outputs);

    if (i < n_outputs)
    {
        // update weights
        float local_gradient = outputs[i] - expect[i];
        weights1[i][n_hidden] -= local_gradient;
        for (size_t j = 0; j < n_hidden; j++)
            weights1[i][j] -= local_gradient * intermediates[j];
    }

    if (i < n_hidden)
    {
        // compute local gradient, δ_i = ϕ'(v_i) × ∑ δ_j w_ji
        float local_gradient = 0.f;
        for (size_t j = 0; j < n_outputs; j++)
        {
            float local_gradient_j = outputs[j] - expect[j];
            local_gradient += weights1[j][i] * local_gradient_j;
        }
        local_gradient *= sigmoid_prime(local_fields[i]);

        // update weights
        weights0[n_inputs][i] -= local_gradient;
        for (size_t j = 0; j < n_inputs; j++)
            weights0[j][i] -= local_gradient * inputs[j];
    }
}

__global__ void init(int seed)
{
    const int i = threadIdx.x;
    hiprandState_t state;
    hiprand_init(seed, i, 0, &state);
    if (i < n_hidden)
    {
        for (size_t j = 0; j < n_inputs; j++)
            weights0[j][i] = 2.f * hiprand_uniform(&state) - 1.f;
    }
    if (i < n_outputs)
    {
        for (size_t j = 0; j < n_hidden; j++)
            weights1[i][j] = 2.f * hiprand_uniform(&state) - 1.f;
    }
}

__global__ void do_compute(float* inputs, float* outputs)
{
    compute(inputs, outputs);
}

__global__ void do_train(float* inputs, float* outputs)
{
    train(inputs, outputs);
}

void import_case(mnist_t* mnist, float* input, float* expect)
{
    // get data
    unsigned char image[mnist->n_pixels];
    unsigned int label = mnist_next(mnist, image);

    // set input
    for (size_t i = 0; i < mnist->n_pixels; i++)
        input[i] = image[i] / 256.f;

    // set expected output
    for (size_t i = 0; i < n_outputs; i++)
        expect[i] = 0.f;
    expect[label] = 1.f;
}

int main()
{
    float hinputs[n_inputs];
    float hexpect[n_outputs];
    float houtput[n_outputs];

    float* dinputs; hipMalloc((void**)&dinputs,  n_inputs*sizeof(float));
    float* dexpect; hipMalloc((void**)&dexpect, n_outputs*sizeof(float));
    float* doutput; hipMalloc((void**)&doutput, n_outputs*sizeof(float));

    size_t n_nodes = max(n_outputs, n_hidden);
    init<<<1, n_nodes>>>(time(NULL));

    printf("training\n");
    mnist_t mnist;
    mnist_init(&mnist, "mnist/train-labels-idx1-ubyte", "mnist/train-images-idx3-ubyte");
    for (int i = 0; i < mnist.n_elements; i++)
    {
        import_case(&mnist, hinputs, hexpect);

        hipMemcpy(dinputs, hinputs,  n_inputs*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(dexpect, hexpect, n_outputs*sizeof(float), hipMemcpyHostToDevice);
        do_train<<<1, n_nodes>>>(dinputs, dexpect);
    }
    mnist_exit(&mnist);

    printf("testing\n");
    mnist_init(&mnist, "mnist/t10k-labels-idx1-ubyte", "mnist/t10k-images-idx3-ubyte");
    size_t classifications[n_outputs][n_outputs] = {{0}};

    for (int i = 0; i < mnist.n_elements; i++)
    {
        import_case(&mnist, hinputs, hexpect);
        hipMemcpy(dinputs, hinputs,  n_inputs*sizeof(float), hipMemcpyHostToDevice);
        do_compute<<<1, n_nodes>>>(dinputs, doutput);
        hipMemcpy(houtput, doutput, n_outputs*sizeof(float), hipMemcpyDeviceToHost);

        // retrieve original label
        int label = 0;
        for (; hexpect[label] != 1.f; label++);

        // get result
        float best = 0;
        int selected = 0;
        for (size_t i = 0; i < n_outputs; i++)
        {
            if (houtput[i] > best)
            {
                best = houtput[i];
                selected = i;
            }
        }

        // log classification
        classifications[label][selected]++;
    }

    // table header
    printf("    ");
    for (size_t j = 0; j < n_outputs; j++)
        printf("%4zu ", j);
    printf(" total\n");
    printf("\n");

    for (size_t i = 0; i < n_outputs; i++)
    {
        size_t total = 0;
        printf("%zu   ", i);
        for (size_t j = 0; j < n_outputs; j++)
        {
            printf("%4zu ", classifications[i][j]);
            total += classifications[i][j];
        }
        printf("  %4zu\n", total);
    }

    printf("\n");
    printf("tot ");
    for (size_t j = 0; j < n_outputs; j++)
    {
        size_t total = 0;
        for (size_t i = 0; i < n_outputs; i++)
            total += classifications[i][j];
        printf("%4zu ", total);
    }
    printf("\n");

    printf("\n");
    printf("Caption: image with digit LINE was classified as a COLUMN digit CELL times\n");

    printf("\n");
    size_t correct = 0;
    size_t total = 0;
    for (size_t i = 0; i < n_outputs; i++)
    {
        for (size_t j = 0; j < n_outputs; j++)
            total += classifications[i][j];
        correct += classifications[i][i];
    }
    printf("%zu / %zu → %5.2f\n", correct, total, 100.f*correct/(float)total);


    hipFree(doutput);
    hipFree(dexpect);
    hipFree(dinputs);
    return 0;
}
